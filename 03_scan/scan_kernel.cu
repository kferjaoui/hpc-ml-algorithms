#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cuda/std/bit>
using cuda::std::bit_ceil;
using cuda::std::countr_zero;

__device__ inline int      ilog2_pow2(uint32_t n) { return countr_zero(n); } 

__device__ inline uint32_t next_pow2_u32(uint32_t n) { return bit_ceil(n); } // ceil to next power of two

__global__
void blelloch_exclusive_scan_singleblock(const int* __restrict__ in_dev,
                                        int* __restrict__ out_dev,
                                        uint32_t N)
{
    // Let's assume one block i.e. gridDim.x is 1 and blockIdx.x is 0
    const uint32_t nextPow2_N = next_pow2_u32(N ? N : 1u);
    const uint32_t depth = ilog2_pow2_u32(nextPow2_N);
    const uint32_t nThreads = blockDim.x; 
    const uint32_t tid = threadIdx.x;  // works as both local and globak index as blockIdx.x = 0

    extern __shared__ int sh[]; //assume that the kernel is launched with enough shared memory allocated i.e. size >= nextPow2_N * sizeof(int)
    // load input array into shared memory (strided here unecessary when nThreads >= nextPow2_N)
    for(size_t idx = tid; idx < nextPow2_N; idx += nThreads){
        sh[idx] = (idx < N) ? in_dev[idx] : 0;
    }
    __syncthreads();

    // upsweep phase
    for(uint32_t d = 0; d<depth-1; d++){
        // only active each second thread (d=0), then each 4th thread (d=1), then each 8th (d=2) and so on...
        // Assumes nThreads >= nextPow2_N
        if( ((tid+1) % (1u<<(d+1)) == 0) && (tid < nextPow2_N) ){
            sh[tid] = sh[tid] + sh[tid-(1u<<d)];
        }

        __syncthreads();
    }

    // clear last element
    if(tid == 0) sh[nextPow2_N-1] = 0;
    __syncthreads();

    // downsweep phase
    for(int d = depth-1; d>=0; d--){
        // E.g N = 16 -> only active each 16th thread (d=depth-1), then each 8th thread (d=depth-2), then each 4th (d=depth-3) and so on...
        if( ((tid+1) % (1u<<(d+1)) == 0) && (tid < nextPow2_N) ){
            int t = sh[tid-(1u<<d)];
            sh[tid-(1u<<d)] = sh[tid];
            sh[tid] = sh[tid] + t;
        }
        __syncthreads();
    }

    // write results to output array
    for(uint32_t idx = tid; idx < N; idx += nThreads){
        out_dev[idx] = sh[idx]; 
    }

}