#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstddef>
#include "dot_kernels.cuh"

__global__
void kernel_small_dot_product(const double* x,
                              const double* y,
                              size_t n, 
                              double* result)
{
    // NOTE: this “small” kernel assumes a single block launch (gridDim.x == 1).
    // It also assumes that blockDim.x is a power of two and at least 64.

    // Per-thread sums
    double sum = 0.0;
    for (size_t idx = threadIdx.x; idx<n; idx+=blockDim.x){
        sum = std::fma(x[idx], y[idx], sum);
    }

    extern __shared__ double sh[];
    sh[threadIdx.x] = sum;
    __syncthreads();

    // block-level reduction
    for(size_t stride=blockDim.x >> 1; stride>= warpSize; stride>>= 1){       //warpSize = 32
        if (threadIdx.x < stride) sh[threadIdx.x] += sh[threadIdx.x + stride]; 
        __syncthreads();
    }

    // warp-level reduction
    double value = sh[threadIdx.x]; // value in threads's register, not in shared memory

    if(threadIdx.x < warpSize){    // threads with indices: 0..31
        unsigned mask = __activemask();  // lanes currently active at this instruction
        #pragma unroll
        for(size_t offset=warpSize/2; offset>0; offset>>= 1 ){ //offset starts as 16 then 8, 4, 2 and 1 
            if (threadIdx.x < offset) value += __shfl_down_sync(mask, value, offset) ; 
        }
    }
    if (threadIdx.x == 0) *result = value;
}