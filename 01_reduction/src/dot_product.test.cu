#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <cmath>
#include <algorithm>

#include "helpers.h"
#include "dot_kernels.cuh"

int main() {
    const size_t n = 1u << 20; // 1,048,576 elements

    // host data
    std::vector<double> hx(n), hy(n);
    for (size_t i = 0; i < n; ++i) {
        // deterministic values (not too large)
        hx[i] = 1.0 / double(i + 1);
        hy[i] = std::sin(0.001 * double(i));
    }

    // CPU reference
    double ref = 0.0;
    for (size_t i = 0; i < n; ++i) ref += hx[i] * hy[i];

    // device memory
    double *dx = nullptr, *dy = nullptr, *dout = nullptr;
    CUDA_CHECK(hipMalloc(&dx, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dy, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dout, sizeof(double)));

    CUDA_CHECK(hipMemcpy(dx, hx.data(), n * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dy, hy.data(), n * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(dout, 0, sizeof(double)));

    // launch config
    hipDeviceProp_t prop{};
    int dev = 0;
    CUDA_CHECK(hipGetDevice(&dev));
    CUDA_CHECK(hipGetDeviceProperties(&prop, dev));

    const int block = 256;
    const dim3 grid(4, 1, 1);
    const size_t shmemBytes = block * sizeof(double);

    // launch
    if (grid.x * grid.y * grid.z == 1){
        dot64_singleblock_warp_downsweep<<<grid, block, shmemBytes>>>(dx, dy, n, dout);
    } else{
        dot64_multiblock_warp_downsweep<<<grid, block, shmemBytes>>>(dx, dy, n, dout);
    }
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // fetch result
    double gpu = 0.0;
    CUDA_CHECK(hipMemcpy(&gpu, dout, sizeof(double), hipMemcpyDeviceToHost));

    // report
    double abs_err = std::abs(gpu - ref);
    double rel_err = abs_err / (std::abs(ref) + 1e-18);

    std::printf("CPU ref : %.17g\n", ref);
    std::printf("GPU dot : %.17g\n", gpu);
    std::printf("abs err : %.3e, rel err : %.3e\n", abs_err, rel_err);

    // cleanup
    hipFree(dx);
    hipFree(dy);
    hipFree(dout);
    return 0;
}
